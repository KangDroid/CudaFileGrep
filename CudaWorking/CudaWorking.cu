#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include "hip/hip_runtime.h"
#include ""

using namespace std;

#define LINE_LENGTH 20 // for now, fix it;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

struct returnStructure {
    int index;
    int lines;
    int isSet;
};

__device__ int devStrcmp(char* comp, char* comp_two, int lsize, int rsize) {

    if (lsize != rsize) {
        return 0;
    }

    for (int a = 0; a < lsize; a++) {
        if (comp[a] != comp_two[a]) {
            return 0;
        }
    }
    return 1;
}

__global__ void compareString(char* ptr, char* compString, int* compare_length, returnStructure *rs, int* limit) {
    char arr_tmp[3];
    int block_id_index = (blockDim.x * blockIdx.x + threadIdx.x);
    //printf("st_Array: %d\n", block_id_index);
    if (block_id_index >= *limit) return;
    int st_array = block_id_index * LINE_LENGTH; // To use thread and block more efficiently, use x/y + thread idx
    int length;

    // The Process goes on - Making Array
    for (int i = 0; i < LINE_LENGTH; i++, st_array++) {
        if (ptr[st_array] != 0) {
            arr_tmp[i] = ptr[st_array];
        } else {
            arr_tmp[i] = 0;
            length = i;
            break;
        }
    }
    if (!devStrcmp(arr_tmp, compString, length, *compare_length)) {
        //printf("Not Same!\n");
    } else {
        rs[block_id_index].index = st_array;
        rs[block_id_index].lines = block_id_index;
        rs[block_id_index].isSet = 1;
    }
}

__global__ void printArray(char* ptr, int* ctr) {
    char arr_tmp[3];
    int st_array = (blockDim.x * blockIdx.x + threadIdx.x) * LINE_LENGTH; // To use thread and block more efficiently, use x/y + thread idx
    //printf("st_Array: %d\n", st_array);

    // The Process goes on - Making Array
    for (int i = 0; i < LINE_LENGTH; i++, st_array++) {
        if (ptr[st_array] != 0) {
            arr_tmp[i] = ptr[st_array];
        } else {
            arr_tmp[i] = 0;
            break;
        }
    }
    //printf("The Index: %d and array index is: %d\n", st_array, (blockDim.x * blockIdx.x + threadIdx.x));
    ctr[(blockDim.x * blockIdx.x + threadIdx.x)] = 10;
}

int main(void) {

    // Host Constant variables.
    const int LN_LIMIT_PER_BLOCK = 1024;

    // Host Variable for Cuda Iteration.(Block Count)
    int BLOCK_CTR;

    // Host-Related variables
    char* dev_Array;
    char** read_input = nullptr;
    int line_ctr;
    std::string compStringVar = "90";
    int stringValLength = compStringVar.length();
    int* returnArray;
    struct returnStructure* rs;

    // Device-Related variables
    char* real_dev;
    char* dev_com_string;
    int* compStringLength;
    int* test_array;
    int* limit_exceed_chk;
    struct returnStructure* dev_rs;

    // The Host Code
    std::string tmp;
    std::ifstream fst("C:\\Users\\KangDroid\\Desktop\\test.txt");
    std::ifstream if_ctr("C:\\Users\\KangDroid\\Desktop\\test.txt");
    line_ctr = std::count(std::istreambuf_iterator<char>(if_ctr),
        std::istreambuf_iterator<char>(), '\n');
    printf("Line count: %d\n", line_ctr);

    // Calculate how many iterations we need to act.
    BLOCK_CTR = line_ctr / LN_LIMIT_PER_BLOCK;
    if ((line_ctr % LN_LIMIT_PER_BLOCK) != 0) {
        BLOCK_CTR++;
    } // And line_ctr is the limit.

    // No init the thing.
    returnArray = new int[line_ctr];
    rs = new returnStructure[line_ctr];

    read_input = new char* [line_ctr];
    for (int i = 0; i < line_ctr; i++) {
        fst >> tmp;
        read_input[i] = new char[LINE_LENGTH]; // The Null value
        strcpy(read_input[i], tmp.c_str());
        //printf("%s\n", read_input[i]);
    }

    int ctr_devarr = line_ctr * LINE_LENGTH;
    dev_Array = new char[ctr_devarr];
    int ctr_arr_aux = 0;

    // Copy it to First Dim Array;
    for (int i = 0; i < line_ctr; i++) {
        for (int a = 0; a < LINE_LENGTH; a++) {
            dev_Array[ctr_arr_aux] = read_input[i][a];
            ctr_arr_aux++;
        }
    }

    // Cuda Kernel Call
    gpuErrchk(hipMalloc((void**)&real_dev, sizeof(char) * ctr_devarr));
    gpuErrchk(hipMalloc((void**)&dev_com_string, sizeof(char) * compStringVar.length()));
    gpuErrchk(hipMalloc((void**)&compStringLength, sizeof(int)));
    gpuErrchk(hipMalloc((void**)&test_array, sizeof(int) * line_ctr));
    gpuErrchk(hipMalloc((void**)&dev_rs, sizeof(struct returnStructure) * line_ctr));
    gpuErrchk(hipMalloc((void**)&limit_exceed_chk, sizeof(int)));
    gpuErrchk(hipMemcpy(limit_exceed_chk, &line_ctr, sizeof(int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dev_com_string, compStringVar.c_str(), sizeof(char) * compStringVar.length(), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(compStringLength, &stringValLength, sizeof(int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(real_dev, dev_Array, sizeof(char) * ctr_devarr, hipMemcpyHostToDevice));

    printf("Block: %d\n", BLOCK_CTR);
    compareString << <BLOCK_CTR, LN_LIMIT_PER_BLOCK >> > (real_dev, dev_com_string, compStringLength, dev_rs, limit_exceed_chk);
    //printArray << <20, 512 >> > (real_dev, test_array);
    gpuErrchk(hipMemcpy(returnArray, test_array, sizeof(int) * line_ctr, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(rs, dev_rs, sizeof(struct returnStructure) * line_ctr, hipMemcpyDeviceToHost));
    for (int i = 0; i < line_ctr; i++) {
        if (rs[i].isSet == 1) {
            printf("Lines: %d\n", rs[i].lines + 1);
        }
    }

    // Remove Dynamically allocated memories.
    hipFree(dev_Array);
    hipFree(dev_com_string);
    hipFree(compStringLength);
    hipFree(test_array);
    hipFree(dev_rs);
    hipFree(limit_exceed_chk);

    for (int i = 0; i < line_ctr; i++) {
        delete[] read_input[i];
    }
    delete[] read_input;
    delete[] dev_Array;
    delete[] returnArray;
    delete[] rs;
    return 0;
}